#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <omp.h>

//размер матрицы
#define N 3

#define block_size 1024

__global__ void mult(float* U, float* L, float* D)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    //for(int i=0; i<N; i++)
    while (i < N) {
	for (int j = 0; j < N; j++)
            for (int t = 0; t < N; t++)
                D[i * N + j] += U[i * N + t] * L[t * N + j];

        i += gridDim.x * blockDim.x;
    }
}


__global__ void tr(float* D, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < N) {
	for (int j = 0; j < N; j++) {
		if(i==j) *sum+=D[i * N + j];
	}
	i += gridDim.x * blockDim.x;
    }

}

int main()
{
    float* U, * L, * D, * d_U, * d_L, * d_D;
    float sum=0.0f, *d_sum;
    U = (float*)malloc(N * N * sizeof(float));
    L = (float*)malloc(N * N * sizeof(float));
    D = (float*)malloc(N * N * sizeof(float));
    
    hipMalloc((void**)&d_U, N * N * sizeof(float));
    hipMalloc((void**)&d_L, N * N * sizeof(float));
    hipMalloc((void**)&d_D, N * N * sizeof(float));
    hipMalloc((void**)&d_sum, sizeof(float));
    
    memset(D, 0, N * N * sizeof(float));
    hipMemset(d_D, 0, N * N * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++) 
        {
            if (i > j) U[i * N + j] = 0;
            else U[i * N + j] = -50 + rand() % 100;
        }
    }

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            if (i < j) L[i * N + j] = 0;
            else L[i * N + j] = -50 + rand() % 100;
        }
    }

    printf("U:\n");
    for (int i = 0; i < N; i++) 
    {
        for (int j = 0; j < N; j++) 
        {
            printf("%f\t", U[i*N+j]);
        }
        printf("\n");
    }

    printf("\nL:\n");
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%f\t", L[i * N + j]);
        }
        printf("\n");
    }

    hipMemcpy(d_U, U, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_L, L, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_sum, &sum, sizeof(float), hipMemcpyHostToDevice);

    dim3 block(block_size);
    dim3 grid((N + block_size - 1) / block_size);

    double time_sec = omp_get_wtime();

    mult <<< grid, block >>> (d_U, d_L, d_D);
    hipMemcpy(D, d_D, N * N * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    
    tr <<< grid, block >>> (d_D, d_sum);

    hipMemcpy(&sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);
    time_sec = omp_get_wtime() - time_sec;

    printf("\nMatrix D (D=U*L)\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f\t", D[i*N+j]);
        }
        printf("\n");
    }

    printf("\nThe sum of the diagonal elements of the matrix D = %f\n",sum);

    printf("\nSize = %i\nTime = %f\n", N, time_sec);

    free(U);
    free(L);
    free(D);

    hipFree(d_U);
    hipFree(d_L);
    hipFree(d_D);
    hipFree(d_sum);

    return 0;
}
